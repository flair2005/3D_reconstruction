#include "hip/hip_runtime.h"
/*
 * Software License Agreement (BSD License)
 *
 *  Point Cloud Library (PCL) - www.pointclouds.org
 *  Copyright (c) 2011, Willow Garage, Inc.
 *
 *  All rights reserved.
 *
 *  Redistribution and use in source and binary forms, with or without
 *  modification, are permitted provided that the following conditions
 *  are met:
 *
 *   * Redistributions of source code must retain the above copyright
 *     notice, this list of conditions and the following disclaimer.
 *   * Redistributions in binary form must reproduce the above
 *     copyright notice, this list of conditions and the following
 *     disclaimer in the documentation and/or other materials provided
 *     with the distribution.
 *   * Neither the name of Willow Garage, Inc. nor the names of its
 *     contributors may be used to endorse or promote products derived
 *     from this software without specific prior written permission.
 *
 *  THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS
 *  "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT
 *  LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS
 *  FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE
 *  COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT,
 *  INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
 *  BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 *  LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 *  CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT
 *  LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN
 *  ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 *  POSSIBILITY OF SUCH DAMAGE.
 *
 */

#include "device.hpp"
#include ""
//#include <opencv2/core/core.hpp>
//#include <opencv2/highgui/highgui.hpp>
//#include <opencv2/imgproc/imgproc.hpp>
//#include <opencv2/gpu/gpu.hpp>
//using namespace cv;

using namespace pcl::device;
using namespace pcl::gpu;


namespace pcl
{
  namespace device
  {
    __global__ void
    computeVmapKernel (const PtrStepSz<unsigned short> depth, PtrStep<float> vmap, float fx_inv, float fy_inv, float cx, float cy)
    {
      int u = threadIdx.x + blockIdx.x * blockDim.x;
      int v = threadIdx.y + blockIdx.y * blockDim.y;

      if (u < depth.cols && v < depth.rows)
      {
        float z = depth.ptr (v)[u] / 1000.f; // load and convert: mm -> meters

        if (z != 0)
        {
          float vx = z * (u - cx) * fx_inv;
          float vy = z * (v - cy) * fy_inv;
          float vz = z;

          vmap.ptr (v                 )[u] = vx;
          vmap.ptr (v + depth.rows    )[u] = vy;
          vmap.ptr (v + depth.rows * 2)[u] = vz;
        }
        else
          vmap.ptr (v)[u] = numeric_limits<float>::quiet_NaN ();
	  
	  //printf(" All going well in ComputeVmapKernel ");

      }
    }

    __global__ void
    computeNmapKernel (int rows, int cols, const PtrStep<float> vmap, PtrStep<float> nmap)
    {
      int u = threadIdx.x + blockIdx.x * blockDim.x;
      int v = threadIdx.y + blockIdx.y * blockDim.y;

      if (u >= cols || v >= rows)
        return;

      if (u == cols - 1 || v == rows - 1)
      {
        nmap.ptr (v)[u] = numeric_limits<float>::quiet_NaN ();
        return;
      }

      float3 v00, v01, v10;
      v00.x = vmap.ptr (v  )[u];
      v01.x = vmap.ptr (v  )[u + 1];
      v10.x = vmap.ptr (v + 1)[u];

      if (!isnan (v00.x) && !isnan (v01.x) && !isnan (v10.x))
      {
        v00.y = vmap.ptr (v + rows)[u];
        v01.y = vmap.ptr (v + rows)[u + 1];
        v10.y = vmap.ptr (v + 1 + rows)[u];

        v00.z = vmap.ptr (v + 2 * rows)[u];
        v01.z = vmap.ptr (v + 2 * rows)[u + 1];
        v10.z = vmap.ptr (v + 1 + 2 * rows)[u];

        float3 r = normalized (cross (v01 - v00, v10 - v00));

        nmap.ptr (v       )[u] = r.x;
        nmap.ptr (v + rows)[u] = r.y;
        nmap.ptr (v + 2 * rows)[u] = r.z;
      }
      else
        nmap.ptr (v)[u] = numeric_limits<float>::quiet_NaN ();
	
	//printf(" All going well in ComputeNmapKernel ");
    }
    
    ///////////////////////////////////////////////////////////////////////////////////////////////////////////////
   /* __global__ void
    computeRmapKernel(const PtrStep<float> vmap, PtrStep<float> nmap, const Mat33& Rmat, const float3& tvec, float fx, float fy, int rows, int cols, PtrStep<float> rmap)
    {
      int u = threadIdx.x + blockIdx.x * blockDim.x;
      int v = threadIdx.y + blockIdx.y * blockDim.y;
      
      const float qnan = pcl::device::numeric_limits<float>::quiet_NaN ();
      
      float3 v_temp, v_g = make_float3 (qnan, qnan, qnan);
      
      v_temp.x = vmap.ptr (v)[u];

        if (!isnan (v_temp.x))
        {
          v_temp.y = vmap.ptr (v + rows)[u];
          v_temp.z = vmap.ptr (v + 2 * rows)[u];

          v_g = Rmat * v_temp + tvec;

        }

        //normals
        float3 n_temp, n_g = make_float3 (qnan, qnan, qnan);
        n_temp.x = nmap.ptr (v)[u];

        if (!isnan (n_temp.x))
        {
          n_temp.y = nmap.ptr (v + rows)[u];
          n_temp.z = nmap.ptr (v + 2 * rows)[u];

          n_g = Rmat * n_temp;

        }
     
     if (u < cols && v < rows)
      {
        float z = vmap.ptr (v + 2 * rows)[u]; // load and convert: mm -> meters

        if (z != 0)
        {
          float rx = (1/rsqrtf(2)) * ((v_g.z/fx)/n_g.z);
          float ry = (1/rsqrtf(2)) * ((v_g.z/fy)/n_g.z);
          float rz = z;

          rmap.ptr (v                 )[u] = rx;
          rmap.ptr (v + rows    )[u] = ry;
          rmap.ptr (v + rows * 2)[u] = rz;
        }
        else
          rmap.ptr (v)[u] = numeric_limits<float>::quiet_NaN ();
	  //printf(" All going well in ComputeRmapKernel ");

      }
    }*/
	
  //}
//}
//----------------------------------------------------------------------------------------------------------------------------------
/*Copyright (C) 2016 Multimedia Communication Laboratory - Illinois Institute of Technology, Chicago*/

//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
__global__ void
    computeRegionKernel (int x, int y, const PtrStepSz<unsigned short> depth, PtrStep<unsigned char> dynmap, PtrStep<unsigned short> rgnmap)
	 {
		 
		 int u = threadIdx.x + blockIdx.x * blockDim.x;
         int v = threadIdx.y + blockIdx.y * blockDim.y;
		 
		 //printf("%d ", dynmap.ptr(y)[x]);
		 if ( u > depth.cols || u < 0 || v > depth.rows || v < 0)
			return;
		
		 if (dynmap.ptr(v)[u] != 255)
             rgnmap.ptr(v)[u] = 0;

		 else if (dynmap.ptr(v)[u] == 255)
		{
			//printf("inside first if");
			//down
			if ((abs(depth.ptr(v)[u] - depth.ptr(v+1)[u])<= 3))
				{
				
					dynmap.ptr(v+1)[u] = 255;
					rgnmap.ptr(v+1)[u] = depth.ptr(v+1)[u];
					//printf("in down ");
				}

			//right
			if ((abs(depth.ptr(v)[u] - depth.ptr(v)[u+1])<= 3)) 
				{
				
					dynmap.ptr(v)[u+1] = 255;
					rgnmap.ptr(v)[u+1] = depth.ptr(v)[u+1];
					//printf("in right ");
				
				}

			//up
			if ((abs(depth.ptr(v)[u] - depth.ptr(v-1)[u])<= 3)) 
				{
				
					dynmap.ptr(v-1)[u] = 255;
					rgnmap.ptr(v-1)[u] = depth.ptr(v-1)[u];
					//printf("in up ");
				
				}

			//left
			if ((abs(depth.ptr(v)[u] - depth.ptr(v)[u-1]) <= 3))
				{
				
					dynmap.ptr(v)[u-1] = 255;
					rgnmap.ptr(v)[u-1] = depth.ptr(v)[u-1];
					//printf("in left ");
				
				}
			//printf("in region growing kernel");
		}
	 }

  }
}
///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

__global__ void
	computeEroisonKernel (int rows, int cols, PtrStep<unsigned char> src) 
   {
	   int u = threadIdx.x + blockIdx.x * blockDim.x;
	   int v = threadIdx.y + blockIdx.y * blockDim.y;

	   if ( u >= cols - 1 || v >= rows - 1 || u <= 0 || v <= 0) // excluding first row/col and last row/col
		   return;

	   /*if (src.ptr(v)[u] == 0)
	      {
			  dst.ptr(v)[u] == 0;
	      }*/

	   if (src.ptr(v)[u] == 255 && src.ptr(v-1)[u] == 255 && src.ptr(v+1)[u] == 255 &&
		        src.ptr(v-1)[u-1] == 255 && src.ptr(v)[u-1] == 255 && src.ptr(v+1)[u-1] == 255 && 
		        src.ptr(v-1)[u+1] == 255 && src.ptr(v)[u+1] == 255 && src.ptr(v+1)[u+1] == 255)
	      {
			  src.ptr(v)[u] = 255;
	      }

	   else 
		   src.ptr(v)[u] = 0;
   }



///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
__global__ void
    computeICPmapKernel (int rows, int cols, PtrStep<unsigned char> icpmap)
    {
      int u = threadIdx.x + blockIdx.x * blockDim.x;
      int v = threadIdx.y + blockIdx.y * blockDim.y;

      if (u >= cols || v >= rows)
        return;

      icpmap.ptr(v)[u] = 0;
    }

__global__ void
    computeRMapKernel (int rows, int cols, PtrStep<float> rmap)
    {
      int u = threadIdx.x + blockIdx.x * blockDim.x;
      int v = threadIdx.y + blockIdx.y * blockDim.y;

      if (u >= cols || v >= rows)
        return;

      rmap.ptr(v)[u] = 0.f;
	  //rmap.ptr(v + rows)[u] = 0.f;
	  //rmap.ptr(v + 2 * rows)[u] = 0.f;
    }

__global__ void
    computeCMapKernel (int rows, int cols, PtrStep<float> cmap)
    {
      int u = threadIdx.x + blockIdx.x * blockDim.x;
      int v = threadIdx.y + blockIdx.y * blockDim.y;

      if (u >= cols || v >= rows)
        return;

      cmap.ptr(v)[u] = 0.f;
	  //cmap.ptr(v + rows)[u] = 0.f;
	  //cmap.ptr(v + 2 * rows)[u] = 0.f;
    }

__global__ void
    computeDynMapKernel (int rows, int cols, PtrStep<unsigned char> dynmap)
    {
      int u = threadIdx.x + blockIdx.x * blockDim.x;
      int v = threadIdx.y + blockIdx.y * blockDim.y;

      if (u >= cols || v >= rows)
        return;

      dynmap.ptr(v)[u] = 0;
    }

//--------------------------------------------------------------------------------------------------------------------------------------------------------

//////////////////////////////////////////////////////////////////////////////////////////////////////////////////
void
pcl::device::createVMap (const Intr& intr, const DepthMap& depth, MapArr& vmap)
{
  vmap.create (depth.rows () * 3, depth.cols ());

  dim3 block (32, 8);
  dim3 grid (1, 1, 1);
  grid.x = divUp (depth.cols (), block.x);
  grid.y = divUp (depth.rows (), block.y);

  float fx = intr.fx, cx = intr.cx;
  float fy = intr.fy, cy = intr.cy;
  
  //printf(" All going well in createVmap ");

  computeVmapKernel<<<grid, block>>>(depth, vmap, 1.f / fx, 1.f / fy, cx, cy);
  cudaSafeCall (hipGetLastError ());
}

//////////////////////////////////////////////////////////////////////////////////////////////////////////////////
void
pcl::device::createNMap (const MapArr& vmap, MapArr& nmap)
{
  nmap.create (vmap.rows (), vmap.cols ());

  int rows = vmap.rows () / 3;
  int cols = vmap.cols ();

  dim3 block (32, 8);
  dim3 grid (1, 1, 1);
  grid.x = divUp (cols, block.x);
  grid.y = divUp (rows, block.y);
  
  //printf(" All going well in createNmap ");

  computeNmapKernel<<<grid, block>>>(rows, cols, vmap, nmap);
  cudaSafeCall (hipGetLastError ());
}

//-------------------------------------------------------------------------------------------------------------------------------------------------------------
/*Copyright (C) 2016 Multimedia Communications Laboratory - Illinoins Institute of Technology, Chicago*/

// modification for radius map computation
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
void
pcl::device::createRMap(const DepthMap& depth, MapArr& rmap)
{
  rmap.create (depth.rows () * 3, depth.cols ());

  /*int rows = depth.rows() * 3;
  int cols = depth.cols ();

  dim3 block (32, 8);
  dim3 grid (1, 1, 1);
  grid.x = divUp (cols, block.x);
  grid.y = divUp (rows, block.y);

  computeRMapKernel<<<grid, block>>>(rows, cols, rmap);
  cudaSafeCall (hipGetLastError ());*/
}

void
pcl::device::createIcpMap(const DepthMap& depth, ICPStatusMap& icpmap)
{
  icpmap.create (depth.rows () , depth.cols ());

  int rows = depth.rows();
  int cols = depth.cols ();

  dim3 block (32, 8);
  dim3 grid (1, 1, 1);
  grid.x = divUp (cols, block.x);
  grid.y = divUp (rows, block.y);

  computeICPmapKernel<<<grid, block>>>(rows, cols, icpmap);
  cudaSafeCall (hipGetLastError ());
}
  

void
pcl::device::createDynMap(const DepthMap& depth, DynamicMap& dynmap)
{
  dynmap.create (depth.rows () , depth.cols ());

  /*int rows = depth.rows();
  int cols = depth.cols ();

  dim3 block (32, 8);
  dim3 grid (1, 1, 1);
  grid.x = divUp (cols, block.x);
  grid.y = divUp (rows, block.y);

  computeDynMapKernel<<<grid, block>>>(rows, cols, dynmap);
  cudaSafeCall (hipGetLastError ());*/
}

void
pcl::device::createRgnMap(const DepthMap& depth, DepthMap& rgnmap)
{
  rgnmap.create (depth.rows () , depth.cols ());

  /*int rows = depth.rows();
  int cols = depth.cols ();

  dim3 block (32, 8);
  dim3 grid (1, 1, 1);
  grid.x = divUp (cols, block.x);
  grid.y = divUp (rows, block.y);

  computeDynMapKernel<<<grid, block>>>(rows, cols, dynmap);
  cudaSafeCall (hipGetLastError ());*/
}
  

void
pcl::device::createCMap(const DepthMap& depth, MapArr& cmap)
{
	cmap.create(depth.rows () * 3, depth.cols ());

	/*int rows = depth.rows() * 3;
    int cols = depth.cols ();

    dim3 block (32, 8);
    dim3 grid (1, 1, 1);
    grid.x = divUp (cols, block.x);
    grid.y = divUp (rows, block.y);

    computeCMapKernel<<<grid, block>>>(rows, cols, cmap);
    cudaSafeCall (hipGetLastError ());*/
}

void
pcl::device::RegionGrowing(const DepthMap& depth, DynamicMap& dynmap, DepthMap& rgnmap)
{
	//cmap.create(depth.rows () * 3, depth.cols ());

	int rows = depth.rows();
    int cols = depth.cols ();

    dim3 block (32, 8);
    dim3 grid (1, 1, 1);
    grid.x = divUp (cols, block.x);
    grid.y = divUp (rows, block.y);
	// printf("in region");

    computeRegionKernel<<<grid, block>>>(rows, cols, depth, dynmap, rgnmap);
    cudaSafeCall (hipGetLastError ());
}
 
void
pcl::device::erosion(const DynamicMap& depth_ero)
{
	int rows = depth_ero.rows();
	int cols = depth_ero.cols();

	dim3 block (32, 8);
	dim3 grid (1,1,1);
	grid.x = divUp (cols, block.x);
	grid.y = divUp (rows, block.y);

	computeEroisonKernel<<<grid, block>>>(rows, cols, depth_ero);
    cudaSafeCall (hipGetLastError());
}

//-------------------------------------------------------------------------------------------------------------------------------------------

namespace pcl
{
  namespace device
  {
    __global__ void
    tranformMapsKernel (int rows, int cols, const PtrStep<float> vmap_src, const PtrStep<float> nmap_src,
                        const Mat33 Rmat, const float3 tvec, PtrStepSz<float> vmap_dst, PtrStep<float> nmap_dst)
    {
      int x = threadIdx.x + blockIdx.x * blockDim.x;
      int y = threadIdx.y + blockIdx.y * blockDim.y;

      const float qnan = pcl::device::numeric_limits<float>::quiet_NaN ();

      if (x < cols && y < rows)
      {
        //vetexes
        float3 vsrc, vdst = make_float3 (qnan, qnan, qnan);
        vsrc.x = vmap_src.ptr (y)[x];

        if (!isnan (vsrc.x))
        {
          vsrc.y = vmap_src.ptr (y + rows)[x];
          vsrc.z = vmap_src.ptr (y + 2 * rows)[x];

          vdst = Rmat * vsrc + tvec;

          vmap_dst.ptr (y + rows)[x] = vdst.y;
          vmap_dst.ptr (y + 2 * rows)[x] = vdst.z;
        }

        vmap_dst.ptr (y)[x] = vdst.x;

        //normals
        float3 nsrc, ndst = make_float3 (qnan, qnan, qnan);
        nsrc.x = nmap_src.ptr (y)[x];

        if (!isnan (nsrc.x))
        {
          nsrc.y = nmap_src.ptr (y + rows)[x];
          nsrc.z = nmap_src.ptr (y + 2 * rows)[x];

          ndst = Rmat * nsrc;

          nmap_dst.ptr (y + rows)[x] = ndst.y;
          nmap_dst.ptr (y + 2 * rows)[x] = ndst.z;
        }

        nmap_dst.ptr (y)[x] = ndst.x;
      }
    }
  }
}

//////////////////////////////////////////////////////////////////////////////////////////////////////////////////
void
pcl::device::tranformMaps (const MapArr& vmap_src, const MapArr& nmap_src,
                           const Mat33& Rmat, const float3& tvec,
                           MapArr& vmap_dst, MapArr& nmap_dst)
{
  int cols = vmap_src.cols ();
  int rows = vmap_src.rows () / 3;

  vmap_dst.create (rows * 3, cols);
  nmap_dst.create (rows * 3, cols);

  dim3 block (32, 8);
  dim3 grid (1, 1, 1);
  grid.x = divUp (cols, block.x);
  grid.y = divUp (rows, block.y);

  tranformMapsKernel<<<grid, block>>>(rows, cols, vmap_src, nmap_src, Rmat, tvec, vmap_dst, nmap_dst);
  cudaSafeCall (hipGetLastError ());
  //printf(" All going well in transformMaps ");

  cudaSafeCall (hipDeviceSynchronize ());
}

namespace pcl
{
  namespace device
  {
    template<bool normalize>
    __global__ void
    resizeMapKernel (int drows, int dcols, int srows, const PtrStep<float> input, PtrStep<float> output)
    {
      int x = threadIdx.x + blockIdx.x * blockDim.x;
      int y = threadIdx.y + blockIdx.y * blockDim.y;

      if (x >= dcols || y >= drows)
        return;

      const float qnan = numeric_limits<float>::quiet_NaN ();

      int xs = x * 2;
      int ys = y * 2;

      float x00 = input.ptr (ys + 0)[xs + 0];
      float x01 = input.ptr (ys + 0)[xs + 1];
      float x10 = input.ptr (ys + 1)[xs + 0];
      float x11 = input.ptr (ys + 1)[xs + 1];

      if (isnan (x00) || isnan (x01) || isnan (x10) || isnan (x11))
      {
        output.ptr (y)[x] = qnan;
        return;
      }
      else
      {
        float3 n;

        n.x = (x00 + x01 + x10 + x11) / 4;

        float y00 = input.ptr (ys + srows + 0)[xs + 0];
        float y01 = input.ptr (ys + srows + 0)[xs + 1];
        float y10 = input.ptr (ys + srows + 1)[xs + 0];
        float y11 = input.ptr (ys + srows + 1)[xs + 1];

        n.y = (y00 + y01 + y10 + y11) / 4;

        float z00 = input.ptr (ys + 2 * srows + 0)[xs + 0];
        float z01 = input.ptr (ys + 2 * srows + 0)[xs + 1];
        float z10 = input.ptr (ys + 2 * srows + 1)[xs + 0];
        float z11 = input.ptr (ys + 2 * srows + 1)[xs + 1];

        n.z = (z00 + z01 + z10 + z11) / 4;

        if (normalize)
          n = normalized (n);

        output.ptr (y        )[x] = n.x;
        output.ptr (y + drows)[x] = n.y;
        output.ptr (y + 2 * drows)[x] = n.z;
      }
    }

    template<bool normalize>
    void
    resizeMap (const MapArr& input, MapArr& output)
    {
      int in_cols = input.cols ();
      int in_rows = input.rows () / 3;

      int out_cols = in_cols / 2;
      int out_rows = in_rows / 2;

      output.create (out_rows * 3, out_cols);

      dim3 block (32, 8);
      dim3 grid (divUp (out_cols, block.x), divUp (out_rows, block.y));
      resizeMapKernel<normalize><< < grid, block>>>(out_rows, out_cols, in_rows, input, output);
      cudaSafeCall ( hipGetLastError () );
      cudaSafeCall (hipDeviceSynchronize ());
    }
  }
}

//////////////////////////////////////////////////////////////////////////////////////////////////////////////////
void
pcl::device::resizeVMap (const MapArr& input, MapArr& output)
{
  resizeMap<false>(input, output);
}

//////////////////////////////////////////////////////////////////////////////////////////////////////////////////
void
pcl::device::resizeNMap (const MapArr& input, MapArr& output)
{
  resizeMap<true>(input, output);
}

namespace pcl
{
  namespace device
  {

    template<typename T>
    __global__ void
    convertMapKernel (int rows, int cols, const PtrStep<float> map, PtrStep<T> output)
    {
      int x = threadIdx.x + blockIdx.x * blockDim.x;
      int y = threadIdx.y + blockIdx.y * blockDim.y;

      if (x >= cols || y >= rows)
        return;

      const float qnan = numeric_limits<float>::quiet_NaN ();

      T t;
      t.x = map.ptr (y)[x];
      if (!isnan (t.x))
      {
        t.y = map.ptr (y + rows)[x];
        t.z = map.ptr (y + 2 * rows)[x];
      }
      else
        t.y = t.z = qnan;

      output.ptr (y)[x] = t;
    }
  }
}

//////////////////////////////////////////////////////////////////////////////////////////////////////////////////
template<typename T> void
pcl::device::convert (const MapArr& vmap, DeviceArray2D<T>& output)
{
  int cols = vmap.cols ();
  int rows = vmap.rows () / 3;

  output.create (rows, cols);

  dim3 block (32, 8);
  dim3 grid (divUp (cols, block.x), divUp (rows, block.y));

  convertMapKernel<T><< < grid, block>>>(rows, cols, vmap, output);
  cudaSafeCall ( hipGetLastError () );
  cudaSafeCall (hipDeviceSynchronize ());
}

template void pcl::device::convert (const MapArr& vmap, DeviceArray2D<float4>& output);
template void pcl::device::convert (const MapArr& vmap, DeviceArray2D<float8>& output);

namespace pcl
{
  namespace device
  {
    __global__ void
    mergePointNormalKernel (const float4* cloud, const float8* normals, PtrSz<float12> output)
    {
      int idx = threadIdx.x + blockIdx.x * blockDim.x;

      if (idx < output.size)
      {
        float4 p = cloud[idx];
        float8 n = normals[idx];

        float12 o;
        o.x = p.x;
        o.y = p.y;
        o.z = p.z;

        o.normal_x = n.x;
        o.normal_y = n.y;
        o.normal_z = n.z;

        output.data[idx] = o;
      }
    }
  }
}

void
pcl::device::mergePointNormal (const DeviceArray<float4>& cloud, const DeviceArray<float8>& normals, const DeviceArray<float12>& output)
{
  const int block = 256;
  int total = (int)output.size ();

  mergePointNormalKernel<<<divUp (total, block), block>>>(cloud, normals, output);
  cudaSafeCall ( hipGetLastError () );
  cudaSafeCall (hipDeviceSynchronize ());
}
